#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <chrono> //for the cpu timing
#include <inttypes.h> //To fprint int64
#include <stdio.h>

#define MAX_THREADS 1000

__global__ void collatzKernel(int *c)
{
	int s = blockDim.x*blockIdx.x + threadIdx.x;
	c[s] = 0;
	unsigned i = s;
	int count = 1;
	while (i>1) {
		++count;
		if (i % 2)
			i = i * 3 + 1;
		else
			i /= 2;
	}
	c[s] = count;
}

using namespace std::chrono;

int main()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	const int arraySize = 1000000;
	int *c = new int[arraySize];

	int *dev_c;

	hipMalloc((void**)&dev_c, arraySize * sizeof(int));
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	hipEventRecord(start);
	collatzKernel << <arraySize / MAX_THREADS, MAX_THREADS >> > (dev_c);
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	hipDeviceSynchronize();

	hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_c);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("calc concluded after %f ms.\n", milliseconds);

	int maxCollatzIdx = 0;
	for (int i = 0; i < arraySize; i++) {
		if (c[maxCollatzIdx] <= c[i])
			maxCollatzIdx = i;
	}
	high_resolution_clock::time_point t2 = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(t2 - t1).count();
	printf("calc and search concluded after %" PRId64 "microseconds.\n", duration);
	printf("%d with a %d length chain.\n", maxCollatzIdx, c[maxCollatzIdx]);
	getchar();
	return 0;
}
