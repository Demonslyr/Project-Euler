
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <fstream>
#include <iostream>
#include <vector>

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void myFirstKernel() {

}

__global__ void simpleAddKernel(int *x, int *y, int *z) {
	*z = *x + *y;
}

__global__ void simpleVectorAdd(int *x, int *y, int *z, int n) {//I can just pass that int in if I dont want it back out? Maybe not. A large static data structure may still need to be copied and referenced
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	z[index] = x[index] + y[index];
}

__global__ void appendTwoStrings(char* u, char* v, char* w) {
	int index = threadIdx.x + blockIdx.x*blockDim.x;

}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512
class util {
public:
	static void random_ints(int *a, int n);
	static char** getDictionary(char *filePath);
};

char** getDictionary(char *filePath) {
	vector<vector<string>> wordBag;
	for (int i = 0; i < 16; i++) {
		wordBag.push_back(vector<string>());
	}
	char * dict[16];
	string word;
	ifstream dictFile;
	dictFile.open(filePath);
	
	while (!dictFile.eof()) {
		dictFile >> word;
		wordBag[word.length()].push_back(word);
	}
	dictFile.close();

	for (int i = 0; i < 16; i++) {
		int wordVectorLength = wordBag[i].size();
		char *flattenedArray = (char *)malloc(wordVectorLength*i * sizeof(char));
		for (int j = 0; j < wordVectorLength; j++) {
			for (int k = 0; k < i; k++) {
				flattenedArray[i*j+k] = wordBag[i][j][k];
			}
		}
		dict[i] = flattenedArray;
	}
	
	getchar();
	return dict;
}


void util::random_ints(int* a, int n)
{
	int i;
	for (i = 0; i < n; ++i)
		a[i] = rand() % 100;
}

int main()
{
	char** dictionary = getDictionary("C:\\Users\\DrMur\\dict.txt");
	char* d_dictionary;

	char* encodedStrings;


	//C:\Users\DrMur\dict.txt
    //const int arraySize = 5;
    //const int a[arraySize] = { 1, 2, 3, 4, 5 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50 };
    //int c[arraySize] = { 0 };

	int *x, *y, *z;
	int *d_x, *d_y, *d_z;

	// int size = sizeof(int);
	int size = N * sizeof(int);

	hipMalloc((void**)&d_x, size);
	hipMalloc((void**)&d_y, size);
	hipMalloc((void**)&d_z, size);

	x = (int *)malloc(size); util::random_ints(x, N);
	y = (int *)malloc(size); util::random_ints(y, N);
	z = (int *)malloc(size);
	
	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

	simpleVectorAdd<<<N/ THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(d_x, d_y, d_z,N);

	hipMemcpy(z, d_z, size, hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++) {
		printf("[z%d,%d],", i,z[i]);
	}
	free(x);
	free(y);
	free(z);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);

	//myFirstKernel<<<100,100>>>();
	printf("Hello world!\n");
	getchar();

    //// Add vectors in parallel.
    //cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    //    c[0], c[1], c[2], c[3], c[4]);

    //// cudaDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = cudaDeviceReset();
    //if (cudaStatus != cudaSuccess) {
    //    fprintf(stderr, "cudaDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
